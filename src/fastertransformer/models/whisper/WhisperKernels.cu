#include "hip/hip_runtime.h"
#include "src/fastertransformer/models/whisper/WhisperKernels.h"
#include "src/fastertransformer/utils/cuda_utils.h"

namespace fastertransformer
{
// assume chan % 2 = 0
__global__ void embedSinusoid( float* out
                   , int batch
                   , int length
                   , int chan
                   , int size
                   , int max_time)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < size)
    {   
        int chanHalfIdx = id % (chan/2);
        int chanIdx = id % chan;
        int seqIdx = (id / chan) % length;
        //int batchIdx = id / (chan * length);
        float scaled_time = seqIdx * expf(-logf((float)max_time) 
                                         / ((float) (chan / 2 - 1))
                                         * chanHalfIdx);
        if(chanIdx < chan/2)
        {   out[id] += sinf(scaled_time);}
        else{   out[id] += cosf(scaled_time);}
    }
}

void invokeEmbedSinusoid(Tensor out_tensor, hipStream_t stream, size_t max_time)
{
    int n = (int) out_tensor.size();
    dim3 block, grid;
    block.x = std::min<int>((int) n, 1024);
    grid.x = ceil(((float)n)/1024);
    embedSinusoid<<<grid, block, 0, stream>>>( out_tensor.getPtr<float>()
                   , (int) out_tensor.shape[0]
                   , (int) out_tensor.shape[1]
                   , (int) out_tensor.shape[2]
                   , n
                   , (int) max_time);
}

__global__ void causalAttnMask(float* out, int batch, int length, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n)
    {
        int qIdx = (idx/length)%length;
        int kIdx = idx % length;
        if(qIdx >= kIdx) out[idx] = 1.0f; else out[idx] = 0.0f;
    }
}

__global__ void encoderAttnMask(float* out, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<n) out[idx] = 1.0f; else out[idx] = 0.0f;
}

void invokeEncoderAttnMask(float* out, size_t batch, size_t seq, hipStream_t stream)
{ 
    int n = batch * seq * seq;
    dim3 block,grid;
    block.x = std::min<int>(1024,n);
    grid.x = ceil(((float) n)/1024);
    encoderAttnMask<<<grid,block,0,stream>>>(out,n);
}

void invokeCausalAttnMask(float* out, size_t batch, size_t seq, hipStream_t stream)
{
    int n = batch * seq * seq;
    dim3 block,grid;
    block.x = std::min<int>(1024, n);
    grid.x = ceil(((float) n)/1024);
    causalAttnMask<<<grid,block,0,stream>>>(out, batch, seq, n);

}
}